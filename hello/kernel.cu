#include "hip/hip_runtime.h"
#include "./kernel.h"

#include <cstddef>

// per reference ! --> immer per value
// es wird auf einen anderen speicher gegriffen -> es kracht
// es werden immer ein vielfaches von 32 an Thread gestartet -> bei 33 Zeichen werden 64 Threads gestartet
// deswegen wir auch size mit�bergeben

// thread nummer ist relative zum block
// block nummer ist relativ zur grafikkarte
// daraus muss ide absolute threadnummer berechnet werden

// divergenten code vermeiden! -> eine der gr��ten Bremsen
__global__ void kernel(char * const p_dst, char const * const p_src, std::size_t const size) {
	// blockDim Anzahl der Threads pro block
	auto const t{ blockIdx.x * blockDim.x + threadIdx.x }; // -> absolute Threadnumber

	if (t < size) {
		p_dst[t] = p_src[t];
	}
}

hipError_t call_kernel(dim3 const big, dim3 const tib, char * const p_dst, char const * const p_src, std::size_t const size) {
	// blocks in grid
	// threads in block
	// 3 kernel a 512 threads
	kernel <<<big, tib>>> (p_dst, p_src, size);
	return hipGetLastError();
}